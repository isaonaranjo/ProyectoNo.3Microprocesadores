#include "hip/hip_runtime.h"

/*
|--------------------------------------------------------|
UNIVERSIDAD DEL VALLE DE GUATEMALA
CC3056 - Programación de Microprocesadores

Autores: 
Josue Sagastume  18173
Isabel Ortiz  18176
Mario Perdomo 18029

Fecha: 29/10/2019
Archivo: CUDA_Proyecto
Descripcion: 
Determina el promedio de los datos obtenidos del 
sensor UV GUVA - S12SD en un determinado rango de tiempo.
|--------------------------------------------------------|
*/

// Librerias a utilizar
#include <stdio.h> 
#include <stdlib.h> 
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream> 
#include <bits/stdc++.h> 

//rangos de tiempo a utilizar::
// 6 a 10, 10 a 2 y 2 a 6 

#define N (6250) //Hilos creados por la cantidad de datos por cada stream 
#define THREADS_PER_BLOCK (6250) //Hilos por bloque
#define N_BLOCKS (N/THREADS_PER_BLOCK) // Bloques creados

// Esta funcion sirve para obtener promedios de los datos obtenidos en un rango de tiempo
// IMPORTANTE: no se como haremos para parametrizar el tiempo... - Mario
// Lo deje pendiente para hablar con ustedes - Mario
/*__global__ void operation( int *a, int *b, int *c )
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x; // indice 
	// Solo trabajan N hilos
	if (myID < N)
	{
		c[myID] = a[myID] + b[myID];
	}
}
__global__ void operation2( int *a, int *b, int *c )
{
	// Originalmente no funcionaba, ya que faltaba el Id del bloque a utilizar
	int myID = threadIdx.x + blockDim.x* blockIdx.x;

	// Solo trabajan N hilos
	if (myID < N)
	{
		c[myID] = a[myID] * b[myID];
	}
}
__global__ void operation3( int *a, int *b, int *c )
{
	// Originalmente no funcionaba, ya que faltaba el Id del bloque a utilizar
	int myID = threadIdx.x + blockDim.x* blockIdx.x;

	// Solo trabajan N hilos
	if (myID < N)
	{
		c[myID] = a[myID] * b[myID];
	}
}
__global__ void operation4( int *a, int *b, int *c )
{
	int myID = threadIdx.x + blockDim.x* blockIdx.x;

	// Solo trabajan N hilos
	if (myID < N)
	{
		c[myID] = a[myID] * b[myID];
	}
}
*/

int main(int argc, char** argv)
{
	//Se hace cuatro streams para realizar operaciones asincronas
	hipStream_t stream1, stream2, stream3, stream4;

	//Se instancia un array de strings
	vector <string> data;
	int rayo = 0;
	// Instancia para abrir archivos .txt
	ifstream inFile;
	inFile.open("test.txt");
	//Programación defensiva si no existe
    if (!inFile) {
        cout << "El archivo no fue abierto correctamente... \n";
        exit(1); // termina el programa 

    while (inFile >> x) {	
    	//Agarra todos los string del txt y lo pasa a un vector para luego convertirlo en int o float
    	//fuente: https://www.geeksforgeeks.org/array-strings-c-3-different-ways-create/
    	stringstream uv(x)
    	uv >> y;
    	data.push_back(y);
    }
    // Despues haremos que cada dato se convierte en int o float
    // utilizando este comando: https://www.geeksforgeeks.org/converting-strings-numbers-cc/
    inFile.close();
	
	int *a1, *b1, *c1; 									// stream 1 mem ptrs
	int *a2, *b2, *c2; 									// stream 2 mem ptrs
	int *a3, *b3, *c3; 									// stream 3 mem ptrs
	int *a4, *b4, *c4; 									// stream 4 mem ptrs

	int *dev_a1, *dev_b1, *dev_c1; 						// stream 1 mem ptrs
	int *dev_a2, *dev_b2, *dev_c2; 						// stream 2 mem ptrs
	int *dev_a3, *dev_b3, *dev_c3; 						// stream 3 mem ptrs
	int *dev_a4, *dev_b4, *dev_c4; 						// stream 4 mem ptrs
	
	//stream 1
	hipMalloc( (void**)&dev_a1, N * sizeof(int) );
	hipMalloc( (void**)&dev_b1, N * sizeof(int) );
	hipMalloc( (void**)&dev_c1, N * sizeof(int) );

	hipHostAlloc( (void**)&a1, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&b1, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&c1, N * sizeof(int), hipHostMallocDefault);
	
	//stream 2
	hipMalloc( (void**)&dev_a2, N * sizeof(int) );
	hipMalloc( (void**)&dev_b2, N * sizeof(int) );
	hipMalloc( (void**)&dev_c2, N * sizeof(int) );

	hipHostAlloc( (void**)&a2, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&b2, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&c2, N * sizeof(int), hipHostMallocDefault);

	//Stream 3
	hipMalloc( (void**)&dev_a3, N * sizeof(int) );
	hipMalloc( (void**)&dev_b3, N * sizeof(int) );
	hipMalloc( (void**)&dev_c3, N * sizeof(int) );

	hipHostAlloc( (void**)&a3, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&b3, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&c3, N * sizeof(int), hipHostMallocDefault);

	//Stream 4
	hipMalloc( (void**)&dev_a4, N * sizeof(int) );
	hipMalloc( (void**)&dev_b4, N * sizeof(int) );
	hipMalloc( (void**)&dev_c4, N * sizeof(int) );

	hipHostAlloc( (void**)&a4, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&b4, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&c4, N * sizeof(int), hipHostMallocDefault);

	// Falta definir como se estaran entregando los datos
	for (int i =0; i<N; i++){
		a1[i]= i;
		b1[i]= a1[i] + i;

		a2[i]= i;
		b2[i]= a1[i] * i;

	}

	for(int i=0;i < N;i+= N*2) { // loop over data in chunks
	// interweave stream 1 and steam 2
		// Faltaba los asyncs en la memoria cuda
		hipMemcpyAsync(dev_a1,a1,N*sizeof(int),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_a2,a2,N*sizeof(int),hipMemcpyHostToDevice,stream2);
		hipMemcpyAsync(dev_a3,a3,N*sizeof(int),hipMemcpyHostToDevice,stream3);
		hipMemcpyAsync(dev_a4,a4,N*sizeof(int),hipMemcpyHostToDevice,stream4);
		// Faltaba los asyncs en la memoria cuda

		hipMemcpyAsync(dev_b1,b1,N*sizeof(int),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_b2,b2,N*sizeof(int),hipMemcpyHostToDevice,stream2
		hipMemcpyAsync(dev_b3,b4,N*sizeof(int),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_b3,b4,N*sizeof(int),hipMemcpyHostToDevice,stream2);

		// ceil
		//Convierte en numeros floats o decimales a un numero entero

		operation<<<(int)ceil(N/1024)+1,1024,0,stream1>>>(dev_a1,dev_b1,dev_c1);
		operation2<<<(int)ceil(N/1024)+1,1024,1,stream2>>>(dev_a2,dev_b2,dev_c2);
		operation3<<<(int)ceil(N/1024)+1,1024,2,stream3>>>(dev_a3,dev_b3,dev_c3);
		operation4<<<(int)ceil(N/1024)+1,1024,3,stream4>>>(dev_a4,dev_b4,dev_c4);
		
		hipMemcpyAsync(c1,dev_c1,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
		hipMemcpyAsync(c2,dev_c2,N*sizeof(int),hipMemcpyDeviceToHost,stream2);
		hipMemcpyAsync(c3,dev_c3,N*sizeof(int),hipMemcpyDeviceToHost,stream3);
		hipMemcpyAsync(c4,dev_c4,N*sizeof(int),hipMemcpyDeviceToHost,stream4);
	}

	hipStreamSynchronize(stream1); //Faltaba un synchronize para stream 1
	hipStreamSynchronize(stream2); // wait for stream2 to finish
	hipStreamSynchronize(stream3); // wait for stream3 to finish
	hipStreamSynchronize(stream4); // wait for stream4 to finish
	
	printf("Stream 1 \n");
	printf("a1 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",a1[i]);
	}
	printf("b1 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",b1[i]);
	}
	printf("c1 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",c1[i]);
	}
	printf("Stream 2 \n");
	printf("a2 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",a2[i]);
	}
	printf("b2 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",b2[i]);
	}
	printf("c2 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",c2[i]);
	}

	// Destruye todo los streams
	hipStreamDestroy(stream1); 
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);
	hipStreamDestroy(stream4);

	return 0;

	
}